#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define TILE_SIZE 4 

__global__ void meanFilterSharedKernel(float *inputImageKernel, float *outputImagekernel, int dim)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    __shared__ float sharedmem[(TILE_SIZE+2)][(TILE_SIZE+2)];
    
	bool is_x_left = (threadIdx.x == 0), is_x_right = (threadIdx.x == TILE_SIZE-1);
    bool is_y_top = (threadIdx.y == 0), is_y_bottom = (threadIdx.y == TILE_SIZE-1);

	if(is_x_left)
		sharedmem[threadIdx.x][threadIdx.y+1] = 0;
	else if(is_x_right)
		sharedmem[threadIdx.x + 2][threadIdx.y+1]=0;
	if (is_y_top){
		sharedmem[threadIdx.x+1][threadIdx.y] = 0;
		if(is_x_left)
			sharedmem[threadIdx.x][threadIdx.y] = 0;
		else if(is_x_right)
			sharedmem[threadIdx.x+2][threadIdx.y] = 0;
	}
	else if (is_y_bottom){
		sharedmem[threadIdx.x+1][threadIdx.y+2] = 0;
		if(is_x_right)
			sharedmem[threadIdx.x+2][threadIdx.y+2] = 0;
		else if(is_x_left)
			sharedmem[threadIdx.x][threadIdx.y+2] = 0;
	}

	sharedmem[threadIdx.x+1][threadIdx.y+1] = inputImageKernel[row*dim+col];

	if(is_x_left && (col>0))
		sharedmem[threadIdx.x][threadIdx.y+1] = inputImageKernel[row*dim+(col-1)];
	else if(is_x_right && (col<dim-1))
		sharedmem[threadIdx.x + 2][threadIdx.y+1]= inputImageKernel[row*dim+(col+1)];
	if (is_y_top && (row>0)){
		sharedmem[threadIdx.x+1][threadIdx.y] = inputImageKernel[(row-1)*dim+col];
		if(is_x_left)
			sharedmem[threadIdx.x][threadIdx.y] = inputImageKernel[(row-1)*dim+(col-1)];
		else if(is_x_right )
			sharedmem[threadIdx.x+2][threadIdx.y] = inputImageKernel[(row-1)*dim+(col+1)];
	}
	else if (is_y_bottom && (row<dim-1)){
		sharedmem[threadIdx.x+1][threadIdx.y+2] = inputImageKernel[(row+1)*dim + col];
		if(is_x_right)
			sharedmem[threadIdx.x+2][threadIdx.y+2] = inputImageKernel[(row+1)*dim+(col+1)];
		else if(is_x_left)
			sharedmem[threadIdx.x][threadIdx.y+2] = inputImageKernel[(row+1)*dim+(col-1)];
	}

	__syncthreads();

    float filterVector[9] = {sharedmem[threadIdx.x][threadIdx.y], sharedmem[threadIdx.x+1][threadIdx.y],
                    sharedmem[threadIdx.x+2][threadIdx.y], sharedmem[threadIdx.x][threadIdx.y+1],
                    sharedmem[threadIdx.x+1][threadIdx.y+1], sharedmem[threadIdx.x+2][threadIdx.y+1],
                    sharedmem[threadIdx.x] [threadIdx.y+2], sharedmem[threadIdx.x+1][threadIdx.y+2],
                    sharedmem[threadIdx.x+2][threadIdx.y+2]};

	
	{

    float element = 0.0;
    for (int i = 0; i < 9; i++) {
        element += filterVector[i];
    }
	outputImagekernel[row*dim+col] = element/9.0;
	}
}

bool meanFilterGPU(float* image, float* outputImage){
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	hipError_t status;
	int dim = 1280;

    int size =  dim * dim * sizeof(float);
    
	float *deviceinputimage;
    hipMalloc((void**) &deviceinputimage, size);
    
	status = hipGetLastError();              
	if (status != hipSuccess) {                     
		return false;
    }
    
	hipMemcpy(deviceinputimage, image, size, hipMemcpyHostToDevice);
	status = hipGetLastError();              
	if (status != hipSuccess) {                     
		hipFree(deviceinputimage);
		return false;
	}
    
    float *deviceOutputImage;
    hipMalloc((void**) &deviceOutputImage, size);
    
	dim3 dimBlock(TILE_SIZE, TILE_SIZE);
	dim3 dimGrid((int)ceil((float) dim / (float)TILE_SIZE), (int)ceil((float) dim / (float)TILE_SIZE));
    
    clock_t start_filter = clock();
    meanFilterSharedKernel<<<dimGrid, dimBlock>>>(deviceinputimage, deviceOutputImage, dim);
    clock_t end_filter = clock();

    double time_filter = (double)(end_filter-start_filter)/CLOCKS_PER_SEC;
    
    printf("Image size : %d Window size : %d GPU Filter Time: %f\n", dim, 3 , time_filter);
	
	hipMemcpy(outputImage, deviceOutputImage, size, hipMemcpyDeviceToHost);
	status = hipGetLastError();              
	


if (status != hipSuccess) {                     
		hipFree(deviceinputimage);
		hipFree(deviceOutputImage);
		return false;
	}
	hipFree(deviceinputimage);
	hipFree(deviceOutputImage);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float time = 0;
	hipEventElapsedTime(&time,start,stop);
	return true;
}

int main()
{
    // freopen("filtered640GPU.txt","w",stdout);
    int size = 1280*1280;
    float *img, *filtered_img;
    img = (float *)malloc(size * sizeof(float));
	filtered_img = (float *)malloc(size * sizeof(float));

    FILE* inp;
    inp = fopen("img1280.txt","r");
    char line[6];
    char *endptr;
    int i = 0;
    while(1){
        char r = (char)fgetc(inp);
        int k = 0;
        while(r!='\n' && !feof(inp)){
            line[k++] = r;
            r = (char)fgetc(inp);
        }
        line[k]=0;
        if(feof(inp)){
            break;
        }
        img[i] = strtof(line, &endptr);
        i++;
    }

	meanFilterGPU(img, filtered_img); //GPU call for median Filtering with shared Kernel.
    
    // for (int i = 0; i < size; i++)
	// 	printf ("%f\n", filtered_img[i]);
}